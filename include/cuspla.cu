#include "hip/hip_runtime.h"
#pragma once

// CULA includes
#include <culapack.h>
#include <culapackdevice.h>
#include <cula.h>
#include <culablas.h>
#include <culablasdevice.h>
#include <culastatus.h>


#include <cusp/memory.h>
#include <cusp/array2d.h>


// THRUST includes
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>


#include "cusplautils.h"


namespace cuspla{

// *****************  Eigenvalue problem *****************************
template <typename Array2d, typename Array1d>
culaStatus geev(Array2d& H, Array1d& eigvals, Array2d& eigvects, float, cusp::host_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
    typedef typename Array2d::value_type   ValueType;

    cusp::array1d<ValueType, MemorySpace> eigvals_im(H.num_rows);


	status = culaSgeev('N', 'V', H.num_cols, thrust::raw_pointer_cast(H.values.data()),
					   H.num_rows,
					   thrust::raw_pointer_cast(eigvals.data()),
					   thrust::raw_pointer_cast(eigvals_im.data()),
					   NULL, 1,
					   thrust::raw_pointer_cast(eigvects.values.data()), H.num_rows);
    return status;
}

template <typename Array2d, typename Array1d>
culaStatus geev(Array2d& H, Array1d& eigvals, Array2d& eigvects, double, cusp::host_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
    typedef typename Array2d::value_type   ValueType;

    cusp::array1d<ValueType, MemorySpace> eigvals_im(H.num_rows);


	status = culaDgeev('N', 'V', H.num_cols, thrust::raw_pointer_cast(H.values.data()),
					   H.num_cols,
					   thrust::raw_pointer_cast(eigvals.data()),
					   thrust::raw_pointer_cast(eigvals_im.data()),
					   NULL, 1,
					   thrust::raw_pointer_cast(eigvects.values.data()), H.num_rows);
    return status;
}

template <typename Array2d, typename Array1d>
culaStatus geev(Array2d& H, Array1d& eigvals, Array2d& eigvects, float, cusp::device_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
    typedef typename Array2d::value_type   ValueType;

    cusp::array1d<ValueType, MemorySpace> eigvals_im(H.num_rows);

	status = culaDeviceSgeev('N', 'V', H.num_cols, thrust::raw_pointer_cast(H.values.data()),
					   H.num_cols,
					   thrust::raw_pointer_cast(eigvals.data()),
					   thrust::raw_pointer_cast(eigvals_im.data()),
					   NULL, 1,
					   thrust::raw_pointer_cast(eigvects.values.data()), H.num_rows);
    return status;
}

template <typename Array2d, typename Array1d>
culaStatus geev(Array2d& H, Array1d& eigvals, Array2d& eigvects, double, cusp::device_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
    typedef typename Array2d::value_type   ValueType;

    cusp::array1d<ValueType, MemorySpace> eigvals_im(H.num_rows);


	status = culaDeviceDgeev('N', 'V', H.num_cols, thrust::raw_pointer_cast(H.values.data()),
					   H.num_cols,
					   thrust::raw_pointer_cast(eigvals.data()),
					   thrust::raw_pointer_cast(eigvals_im.data()),
					   NULL, 1,
					   thrust::raw_pointer_cast(eigvects.values.data()), H.num_rows);
    return status;
}


// ------------------   Entry point ---------------------
template <typename Array2d, typename Array1d>
culaStatus geev(Array2d& H, Array1d& eigvals, Array2d& eigvects){

	eigvals.resize(H.num_rows);
	eigvects.resize(H.num_rows, H.num_rows);
    return geev(H, eigvals, eigvects, typename Array2d::value_type(), typename Array2d::memory_space(), typename Array2d::orientation());
}




// ***************** Matrix-Matrix multiplication ******************

template <typename ValueType, typename Array2d>
culaStatus gemm(Array2d& A, Array2d& B, Array2d& C, char tA, char tB, ValueType alpha,\
        size_t n, size_t m, size_t k, size_t lda, size_t ldb, \
        float, cusp::host_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
//    typedef typename Array2d::value_type   ValueType;

	status = culaSgemm(tA,tB, n, m, k, alpha,
			thrust::raw_pointer_cast(A.values.data()),
			lda,
			thrust::raw_pointer_cast(B.values.data()),
			ldb,
			ValueType(0),
			thrust::raw_pointer_cast(C.values.data()),
			n);


    return status;
}

template <typename ValueType, typename Array2d>
culaStatus gemm(Array2d& A, Array2d& B, Array2d& C, char tA, char tB, ValueType alpha,\
        size_t n, size_t m, size_t k, size_t lda, size_t ldb, \
        float, cusp::device_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
//    typedef typename Array2d::value_type   ValueType;

	status = culaDeviceSgemm(tA,tB, n, m, k, alpha,
            thrust::raw_pointer_cast(A.values.data()),
            lda,
            thrust::raw_pointer_cast(B.values.data()),
            ldb,
            ValueType(0),
            thrust::raw_pointer_cast(C.values.data()),
            n);


    return status;
}

template <typename ValueType, typename Array2d>
culaStatus gemm(Array2d& A, Array2d& B, Array2d& C, char tA, char tB, ValueType alpha,\
        size_t n, size_t m, size_t k, size_t lda, size_t ldb, \
        double, cusp::host_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
//    typedef typename Array2d::value_type   ValueType;

	status = culaDgemm(tA,tB, n, m, k, alpha,
            thrust::raw_pointer_cast(A.values.data()),
            lda,
            thrust::raw_pointer_cast(B.values.data()),
            ldb,
            ValueType(0),
            thrust::raw_pointer_cast(C.values.data()),
            n);


    return status;
}

template <typename ValueType, typename Array2d>
culaStatus gemm(Array2d& A, Array2d& B, Array2d& C, char tA, char tB, ValueType alpha,\
        size_t n, size_t m, size_t k, size_t lda, size_t ldb, \
        double, cusp::device_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
//    typedef typename Array2d::value_type   ValueType;

	status = culaDeviceDgemm(tA,tB, n, m, k, alpha,
            thrust::raw_pointer_cast(A.values.data()),
            lda,
            thrust::raw_pointer_cast(B.values.data()),
            ldb,
            ValueType(0),
            thrust::raw_pointer_cast(C.values.data()),
            n);


    return status;
}

// --------- Entry point -----------------------
// The upper triangular matrix R will be in H matrix
template <typename Array2d, typename ValueType>
culaStatus gemm(Array2d& A, Array2d& B, Array2d& C, ValueType alpha=ValueType(1),\
		ValueType beta = ValueType(0),  bool transA=false, bool transB=false){
    /*
     * C = alpha*OP(A)*OP(B) + beta*C
     *
     * Default values:
     *   alpha = 1.0
     *   beta = 0.0
     *   transA = transB = false
     *
     * Note: C must be different from both A and B matrix because storing
     *  the result in C will change either matrix A or B going
     *  to change the correct result.
     */


    assert(&A!=&C && &B!=&C);

	char tA = (transA)?'T':'N';
	char tB = (transB)?'T':'N';


    size_t n = A.num_rows;
    size_t kA = A.num_cols;
    size_t lda = n;
    if(transA){
        n = A.num_cols;
        kA = A.num_rows;
        lda = kA;
    }

    size_t m = B.num_cols;
    size_t kB = B.num_rows;
    size_t ldb = kB;
    if(transB){
        m = B.num_rows;
        kB = B.num_cols;
        ldb = m;
    }

    assert(kA == kB);

	C.resize(n,m);
	return gemm(A, B, C, tA, tB, alpha, n, m, kA, lda, ldb,   \
	        typename Array2d::value_type(),   \
	        typename Array2d::memory_space(), \
	        typename Array2d::orientation());
}




// ***************** Matrix-Vector multiplication ******************

template <typename Array2d, typename Array1d>
culaStatus gemv(Array2d& A, Array1d& x, Array1d& y, char tA,\
        float, cusp::host_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
    typedef typename Array2d::value_type   ValueType;

	status = culaSgemv(tA, A.num_rows, A.num_cols, ValueType(1),
			thrust::raw_pointer_cast(A.values.data()),
			A.num_rows,
			thrust::raw_pointer_cast(x.data()),
			ValueType(1),
			ValueType(0),
			thrust::raw_pointer_cast(y.data()),
			ValueType(1));


    return status;
}

template <typename Array2d, typename Array1d>
culaStatus gemv(Array2d& A, Array1d& x, Array1d& y, char tA,\
        float, cusp::device_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
    typedef typename Array2d::value_type   ValueType;

	status = culaDeviceSgemv(tA, A.num_rows, A.num_cols, ValueType(1),
			thrust::raw_pointer_cast(A.values.data()),
			A.num_rows,
			thrust::raw_pointer_cast(x.data()),
			ValueType(1),
			ValueType(0),
			thrust::raw_pointer_cast(y.data()),
			ValueType(1));


    return status;
}

template <typename Array2d, typename Array1d>
culaStatus gemv(Array2d& A, Array1d& x, Array1d& y, char tA,\
        double, cusp::host_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
    typedef typename Array2d::value_type   ValueType;

	status = culaDgemv(tA, A.num_rows, A.num_cols, ValueType(1),
			thrust::raw_pointer_cast(A.values.data()),
			A.num_rows,
			thrust::raw_pointer_cast(x.data()),
			ValueType(1),
			ValueType(0),
			thrust::raw_pointer_cast(y.data()),
			ValueType(1));


    return status;
}

template <typename Array2d, typename Array1d>
culaStatus gemv(Array2d& A, Array1d& x, Array1d& y, char tA,\
        double, cusp::device_memory, cusp::column_major){
    culaStatus status;

    typedef typename Array2d::memory_space MemorySpace;
    typedef typename Array2d::value_type   ValueType;

	status = culaDeviceDgemv(tA, A.num_rows, A.num_cols,ValueType(1),
			thrust::raw_pointer_cast(A.values.data()),
			A.num_rows,
			thrust::raw_pointer_cast(x.data()),
			ValueType(1),
			ValueType(0),
			thrust::raw_pointer_cast(y.data()),
			ValueType(1));


    return status;
}



// ---------------------- Entry point --------------------------
template <typename Array2d, typename Array1d>
culaStatus gemv(Array2d& A, Array1d& x, Array1d& y, bool transA=false){
    /*
    *  y must be different from x.
    */

    assert(&x != &y);
	char tA = (transA)?'T':'N';

    size_t n = A.num_rows;
    size_t m = A.num_cols;
    if(transA){
        n = A.num_cols;
        m = A.num_rows;
    }

    assert(m == x.size());
    y.resize(n);
	return gemv(A, x, y, tA, typename Array2d::value_type(), typename Array2d::memory_space(), typename Array2d::orientation());
}



// *****************  QR Factorization *****************************


// Computes H = I-tau*v*v'
template<typename ValueType, typename Array1d, typename Array2d>
void house_holder(const ValueType tau, const Array1d& v, Array2d& H){

    typedef typename Array2d::value_type   ValueType2;

    size_t N = v.size();
	H.resize(N, N);

    Array2d tmp(N, 1);
    tmp.values = v;
//    thrust::copy(v.begin(), v.end(), tmp.values.begin());

    gemm(tmp, tmp, H, -tau, ValueType2(0), false, true);

    // Adds 1 of each element of the diagonal
    thrust::counting_iterator<int> stencil (0);
    thrust::transform_if(H.values.begin(), H.values.end(), \
            stencil, \
            H.values.begin(), \
            cuspla::plus_const<ValueType2>(ValueType2(1)), \
            cuspla::in_diagonal(N,N));

}




template <typename Array2d, typename Array1d>
culaStatus geqrf(Array2d& A, Array1d& tau, float, cusp::host_memory, cusp::column_major){
	return culaSgeqrf(A.num_rows, A.num_cols, thrust::raw_pointer_cast(A.values.data()),
					   A.num_rows,
					   thrust::raw_pointer_cast(tau.data()));
}

template <typename Array2d, typename Array1d>
culaStatus geqrf(Array2d& A, Array1d& tau, double, cusp::host_memory, cusp::column_major){
	return culaDgeqrf(A.num_rows, A.num_cols, thrust::raw_pointer_cast(A.values.data()),
					   A.num_rows,
					   thrust::raw_pointer_cast(tau.data()));
}

template <typename Array2d, typename Array1d>
culaStatus geqrf(Array2d& A, Array1d& tau, float, cusp::device_memory, cusp::column_major){
	return culaDeviceSgeqrf(A.num_rows, A.num_cols, thrust::raw_pointer_cast(A.values.data()),
					   A.num_rows,
					   thrust::raw_pointer_cast(tau.data()));
}

template <typename Array2d, typename Array1d>
culaStatus geqrf(Array2d& A, Array1d& tau, double, cusp::device_memory, cusp::column_major){
	return culaDeviceDgeqrf(A.num_rows, A.num_cols, thrust::raw_pointer_cast(A.values.data()),
					   A.num_rows,
					   thrust::raw_pointer_cast(tau.data()));
}







// ------------------ Entry point ----------------
// The upper triangular matrix R will be in H matrix
template <typename Array2d>
culaStatus geqrf(Array2d& A, Array2d& Q, Array2d& R, bool get_R=true){


    typedef typename Array2d::memory_space MemorySpace;
    typedef typename Array2d::value_type   ValueType;

    size_t N = A.num_rows;
    size_t M = A.num_cols;
    size_t min_dim = std::min(M,N);

    cusp::array1d<ValueType, MemorySpace> tau(min_dim);

	culaStatus status = geqrf(A, tau, typename Array2d::value_type(), \
	        typename Array2d::memory_space(), \
	        typename Array2d::orientation());


	cusp::array1d<ValueType, MemorySpace> v(N, ValueType(0));
	cusp::array2d<ValueType, MemorySpace, cusp::column_major> H(N,M);

	// Set Q to the identity
	Q.resize(N,N);
	thrust::fill(Q.values.begin(), Q.values.end(), ValueType(0));
    //     Complete the diagonal
    thrust::counting_iterator<int> stencil (0);
    thrust::transform_if(Q.values.begin(), Q.values.end(), \
        stencil, \
        Q.values.begin(), \
        cuspla::assigns<ValueType>(ValueType(1)), \
        cuspla::in_diagonal(N,N));


	// Computes Q = Q*H(k)
	for(size_t k = 0; k<min_dim; k++){
		// define v
		thrust::fill(v.begin(), v.begin()+k, ValueType(0));
		v[k]=ValueType(1);
		thrust::copy(A.values.begin()+(N*k + k+1), A.values.begin()+(N*(k+1)), v.begin()+k+1);



		house_holder(tau[k], v, H);


        Array2d Q_;
		gemm(Q, H, Q_, ValueType(1));
		cusp::copy(Q_, Q);

	}


    if(get_R){
        //computes R is the upper triangular of A
        R.resize(N,M);
        thrust::fill(R.values.begin(), R.values.end(), ValueType(0));
        // Copy the upper triangular of A to R
        thrust::transform_if(A.values.begin(), A.values.end(), \
            thrust::counting_iterator<int>(0), \
            R.values.begin(), \
            cuspla::copy<ValueType>(), \
            cuspla::in_upper_triang(N,M));

    }

	return status;

}




} // end cula namespace

