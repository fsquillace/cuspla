#include "hip/hip_runtime.h"
/*
 * test_cuspla.cu
 *
 *  Created on: Nov 16, 2011
 *      Author: Squillace Filippo
 */

//#define CUSP_USE_TEXTURE_MEMORY


// CUSP includes
#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/print.h>
#include <cusp/multiply.h>
#include <cusp/transpose.h>
#include <cusp/array1d.h>
#include <cusp/array2d.h>
#include <cusp/detail/random.h>


#include <cuspla.cu>


// common
#include <cppunit/ui/text/TestRunner.h>
#include <cppunit/TestFixture.h>
#include <cppunit/TestCaller.h>
#include <cppunit/TestSuite.h>
#include <cppunit/extensions/HelperMacros.h>
#include <cppunit/extensions/TestFactoryRegistry.h>

#include <string.h>
#include <iostream>
#include <stdio.h>
#include <sstream>

void checkStatus(culaStatus status)
{
    if(!status)
        return;
    if(status == culaArgumentError)
        printf("Invalid value for parameter %d\n", culaGetErrorInfo());
    else if(status == culaDataError)
        printf("Data error (%d)\n", culaGetErrorInfo());
    else if(status == culaBlasError)
        printf("Blas error (%d)\n", culaGetErrorInfo());
    else if(status == culaRuntimeError)
        printf("Runtime error (%d)\n", culaGetErrorInfo());
    else
        printf("%s\n", culaGetStatusString(status));

    culaShutdown();
    exit(EXIT_FAILURE);
}



class CusplaTestCase : public CppUnit::TestFixture {

    CPPUNIT_TEST_SUITE (CusplaTestCase);

    CPPUNIT_TEST (test_host_GEMM);
    CPPUNIT_TEST (test_host_transpose_GEMM);
    CPPUNIT_TEST (test_device_GEMM);
    CPPUNIT_TEST (test_device_transpose_GEMM);

    CPPUNIT_TEST (test_host_GEMV);
    CPPUNIT_TEST(test_host_transpose_GEMV);
    CPPUNIT_TEST(test_device_GEMV);
    CPPUNIT_TEST(test_device_traspose_GEMV);

    CPPUNIT_TEST(test_host_GEQRF);
    CPPUNIT_TEST(test_device_GEQRF);

    CPPUNIT_TEST(test_host_GEEV);
    CPPUNIT_TEST(test_device_GEEV);

    CPPUNIT_TEST(test_host_GETRI);
    CPPUNIT_TEST(test_device_GETRI);

    CPPUNIT_TEST_SUITE_END ();

    typedef int    IndexType;
    typedef float ValueType;
    typedef cusp::array2d<ValueType,cusp::device_memory, cusp::column_major> DeviceMatrix_array2d;
    typedef cusp::array2d<ValueType, cusp::host_memory, cusp::column_major>   HostMatrix_array2d;
    typedef cusp::array1d<ValueType,cusp::device_memory> DeviceVector_array1d;
    typedef cusp::array1d<ValueType, cusp::host_memory>   HostVector_array1d;
private:

    std::vector<std::string> path_not_squared;
    std::vector<DeviceMatrix_array2d> dev_mat_not_squared;
    std::vector<HostMatrix_array2d> host_mat_not_squared;


    std::vector<std::string> path_def_pos;
    std::vector<DeviceMatrix_array2d> dev_mat_def_pos;
    std::vector<HostMatrix_array2d> host_mat_def_pos;

public:

  void setUp()
  {

      culaStatus status;
      status = culaInitialize();
      checkStatus(status);


      // ################################ NOT SQUARED #####################
//      path_not_squared = std::vector<std::string>(1);
//      path_not_squared[0] = "data/not-squared/rand90x80.mtx";
      path_not_squared = std::vector<std::string>(5);
      path_not_squared[0] = "data/not-squared/rand9x11.mtx";
      path_not_squared[1] = "data/not-squared/rand10x9.mtx";
      path_not_squared[2] = "data/not-squared/rand90x80.mtx";
      path_not_squared[3] = "data/not-squared/rand90x100.mtx";
      path_not_squared[4] = "data/not-squared/rand100x90.mtx";

      host_mat_not_squared = std::vector<HostMatrix_array2d>(path_not_squared.size());
      dev_mat_not_squared = std::vector<DeviceMatrix_array2d>(path_not_squared.size());
      for(size_t i=0; i<path_not_squared.size(); i++){
          cusp::io::read_matrix_market_file(host_mat_not_squared[i], path_not_squared[i]);
          dev_mat_not_squared[i] = DeviceMatrix_array2d(host_mat_not_squared[i]);
      }


      // ################################ POSITIVE DEFINITE #####################
      path_def_pos = std::vector<std::string>(3);
      path_def_pos[0] = "data/positive-definite/lehmer10.mtx";
      path_def_pos[1] = "data/positive-definite/lehmer50.mtx";
      path_def_pos[2] = "data/positive-definite/lehmer100.mtx";

      host_mat_def_pos = std::vector<HostMatrix_array2d>(path_def_pos.size());
      dev_mat_def_pos = std::vector<DeviceMatrix_array2d>(path_def_pos.size());
      for(size_t i=0; i<path_def_pos.size(); i++){
          cusp::io::read_matrix_market_file(host_mat_def_pos[i], path_def_pos[i]);
          dev_mat_def_pos[i] = DeviceMatrix_array2d(host_mat_def_pos[i]);
      }


  }

  void tearDown()
  {
      culaShutdown();
  }


  void test_host_GEMM()
  {
      for(size_t i=0; i<path_not_squared.size(); i++){
          for(size_t j=0; j<path_not_squared.size(); j++){

              if(host_mat_not_squared[i].num_cols != host_mat_not_squared[j].num_rows)
                  continue;

              HostMatrix_array2d C;
              HostMatrix_array2d C2;

              cuspla::gemm(host_mat_not_squared[i], host_mat_not_squared[j], C,\
            		  ValueType(1),ValueType(0),false,false);
              cusp::multiply(host_mat_not_squared[i], host_mat_not_squared[j], C2);

              ValueType errRel = nrmVector("host_GEMM "+path_not_squared[i]+" "+path_not_squared[j], C.values, C2.values);
              CPPUNIT_ASSERT( errRel < 1.0e-6 );
          }
      }
  }


  void test_host_transpose_GEMM()
  {
      for(size_t i=0; i<path_not_squared.size(); i++){
          for(size_t j=0; j<path_not_squared.size(); j++){

              if(host_mat_not_squared[i].num_rows != host_mat_not_squared[j].num_cols)
                   continue;

              HostMatrix_array2d C;
              HostMatrix_array2d C2, mat_OP1_trans, mat_OP2_trans;

              cuspla::gemm(host_mat_not_squared[i], host_mat_not_squared[j], C,\
            		  ValueType(1),ValueType(0),true, true);

              cusp::transpose(host_mat_not_squared[i], mat_OP1_trans);
              cusp::transpose(host_mat_not_squared[j], mat_OP2_trans);
              cusp::multiply(mat_OP1_trans, mat_OP2_trans, C2);

              ValueType errRel = nrmVector("host_transpose_GEMM "+path_not_squared[i]+" "+path_not_squared[j], C.values, C2.values);
              CPPUNIT_ASSERT( errRel < 1.0e-6 );
          }
      }
  }


  void test_device_GEMM()
  {
      for(size_t i=0; i<path_not_squared.size(); i++){
          for(size_t j=0; j<path_not_squared.size(); j++){

              if(dev_mat_not_squared[i].num_cols != dev_mat_not_squared[j].num_rows)
                                continue;

              DeviceMatrix_array2d C;
              HostMatrix_array2d C2, C_host;

              cuspla::gemm(dev_mat_not_squared[i], dev_mat_not_squared[j], C,\
            		  ValueType(1),ValueType(0),false,false);
              C_host = HostMatrix_array2d(C);

              cusp::multiply(host_mat_not_squared[i], host_mat_not_squared[j], C2);

              ValueType errRel = nrmVector("device_GEMM "+path_not_squared[i]+" "+path_not_squared[j], C_host.values, C2.values);
              CPPUNIT_ASSERT( errRel < 1.0e-6 );
          }
      }
  }


  void test_device_transpose_GEMM()
  {
      for(size_t i=0; i<path_not_squared.size(); i++){
          for(size_t j=0; j<path_not_squared.size(); j++){

              if(dev_mat_not_squared[i].num_rows != dev_mat_not_squared[j].num_cols)
                                 continue;

              DeviceMatrix_array2d C;
              HostMatrix_array2d C2, mat_OP1_trans, mat_OP2_trans, C_host;

              cuspla::gemm(dev_mat_not_squared[i], dev_mat_not_squared[j], C,\
            		  ValueType(1),ValueType(0),true, true);
              C_host = HostMatrix_array2d(C);

              cusp::transpose(host_mat_not_squared[i], mat_OP1_trans);
              cusp::transpose(host_mat_not_squared[j], mat_OP2_trans);
              cusp::multiply(mat_OP1_trans, mat_OP2_trans, C2);

              ValueType errRel = nrmVector("device_transpose_GEMM "+path_not_squared[i]+" "+path_not_squared[j], C_host.values, C2.values);
              CPPUNIT_ASSERT( errRel < 1.0e-6 );

          }
      }
  }


  void test_host_GEMV()
  {
      for(size_t i=0; i<path_not_squared.size(); i++){
          size_t N = host_mat_not_squared[i].num_rows;
          size_t M = host_mat_not_squared[i].num_cols;
          HostVector_array1d vec_OP2 = cusp::detail::random_reals<ValueType>(M);
          HostVector_array1d C(N);
          HostVector_array1d C2(N);

          cuspla::gemv(host_mat_not_squared[i], vec_OP2, C,false);

          cusp::multiply(host_mat_not_squared[i], vec_OP2, C2);

          ValueType errRel = nrmVector("host_GEMV "+path_not_squared[i], C, C2);
          CPPUNIT_ASSERT( errRel < 1.0e-6 );
      }
  }

  void test_host_transpose_GEMV()
  {
      for(size_t i=0; i<path_not_squared.size(); i++){
          size_t N = host_mat_not_squared[i].num_rows;
          size_t M = host_mat_not_squared[i].num_cols;
          HostVector_array1d vec_OP2 = cusp::detail::random_reals<ValueType>(N);
          HostVector_array1d C(M);
          HostVector_array1d C2(M);
          HostMatrix_array2d mat_OP1_trans;

          cuspla::gemv(host_mat_not_squared[i], vec_OP2, C,true);

          cusp::transpose(host_mat_not_squared[i], mat_OP1_trans);
          cusp::multiply(mat_OP1_trans, vec_OP2, C2);

          ValueType errRel = nrmVector("host_transpose_GEMV "+path_not_squared[i], C, C2);
          CPPUNIT_ASSERT( errRel < 1.0e-6 );
    }
  }

  void test_device_GEMV()
  {
      for(size_t i=0; i<path_not_squared.size(); i++){

          size_t N = host_mat_not_squared[i].num_rows;
          size_t M = host_mat_not_squared[i].num_cols;
          DeviceVector_array1d vec_OP2 = cusp::detail::random_reals<ValueType>(M);
          DeviceVector_array1d C(N);
          HostVector_array1d C2(N), vec_OP2_host, C_host;

          cuspla::gemv(dev_mat_not_squared[i], vec_OP2, C,false);

          vec_OP2_host = HostVector_array1d(vec_OP2);
          cusp::multiply(host_mat_not_squared[i], vec_OP2_host, C2);
          C_host = HostVector_array1d(C);

          ValueType errRel = nrmVector("device_GEMV "+path_not_squared[i], C_host, C2);

          CPPUNIT_ASSERT( errRel < 1.0e-6 );

      }
  }

  void test_device_traspose_GEMV()
  {
      for(size_t i=0; i<path_not_squared.size(); i++){

          size_t N = host_mat_not_squared[i].num_rows;
          size_t M = host_mat_not_squared[i].num_cols;
          DeviceVector_array1d vec_OP2 = cusp::detail::random_reals<ValueType>(N);
          DeviceVector_array1d C(M);
          HostVector_array1d C2(M), vec_OP2_host, C_host;

          cuspla::gemv(dev_mat_not_squared[i], vec_OP2, C,true);

          vec_OP2_host = HostVector_array1d(vec_OP2);
          HostMatrix_array2d mat_OP1_trans;
          cusp::transpose(host_mat_not_squared[i], mat_OP1_trans);
          cusp::multiply(mat_OP1_trans, vec_OP2_host, C2);
          C_host = HostVector_array1d(C);


          ValueType errRel = nrmVector("device_traspose_GEMV "+path_not_squared[i], C_host, C2);
          CPPUNIT_ASSERT( errRel < 1.0e-6 );

      }
  }


  void test_host_GEQRF()
  {


      for(size_t i=0; i<path_not_squared.size(); i++){

          size_t n = host_mat_not_squared[i].num_rows;
          size_t m = host_mat_not_squared[i].num_cols;
          HostMatrix_array2d Q(n,n);
          HostMatrix_array2d R(n, m);
          HostMatrix_array2d A(n,m), mat_OP1_copy;

          cusp::copy(host_mat_not_squared[i], mat_OP1_copy);
          cuspla::geqrf(mat_OP1_copy, Q, R, true);

          //Checks orthogonality of Q
          HostMatrix_array2d Qt(n,n), I(n,n), Ip(n,n);
          cusp::transpose(Q, Qt);
          cusp::multiply(Q,Qt, I);
          thrust::fill(Ip.values.begin(), Ip.values.end(), ValueType(0));
          thrust::counting_iterator<int> stencil (0);
          thrust::transform_if(Ip.values.begin(), Ip.values.end(), \
              stencil, \
              Ip.values.begin(), \
              cuspla::assigns<ValueType>(ValueType(1)), \
              cuspla::in_diagonal(n,n));
          ValueType errRel = nrmVector("host_GEQRF orthogonality "+path_not_squared[i], I.values, Ip.values);
          CPPUNIT_ASSERT( errRel < 1.0e-5 );



          // Checks Factorization
          cusp::multiply(Q, R, A);
          errRel = nrmVector("host_GEQRF factorization "+path_not_squared[i], host_mat_not_squared[i].values, A.values);
          CPPUNIT_ASSERT( errRel < 1.0e-5 );

      }
  }

  void test_device_GEQRF()
  {

      for(size_t i=0; i<path_not_squared.size(); i++){
          size_t n = host_mat_not_squared[i].num_rows;
          size_t m = host_mat_not_squared[i].num_cols;
          DeviceMatrix_array2d Q(n,n);
          DeviceMatrix_array2d R(n, m);


          cuspla::geqrf(dev_mat_not_squared[i], Q, R, true);

          //Checks orthogonality of Q
          HostMatrix_array2d I(n,n), Ip(n,n);
          HostMatrix_array2d Q_host,Qt(n,n);
          cusp::copy(Q, Q_host);
          cusp::transpose(Q_host, Qt);
          cusp::multiply(Q_host,Qt, I);
          thrust::fill(Ip.values.begin(), Ip.values.end(), ValueType(0));
          thrust::counting_iterator<int> stencil (0);
          thrust::transform_if(Ip.values.begin(), Ip.values.end(), \
              stencil, \
              Ip.values.begin(), \
              cuspla::assigns<ValueType>(ValueType(1)), \
              cuspla::in_diagonal(n,n));
          ValueType errRel = nrmVector("device_GEQRF orthogonality "+path_not_squared[i], I.values, Ip.values);
          CPPUNIT_ASSERT( errRel < 1.0e-5 );



          // Checks Factorization
          HostMatrix_array2d A(n,m), R_host(n, m);
          cusp::copy(R, R_host);
          cusp::multiply(Q_host, R_host, A);
          errRel = nrmVector("device_GEQRF factorization "+path_not_squared[i], host_mat_not_squared[i].values, A.values);
          CPPUNIT_ASSERT( errRel < 1.0e-5 );

      }
  }



  void test_host_GEEV()
  {

      for(size_t i=0; i<path_def_pos.size(); i++){

          size_t n = host_mat_def_pos[i].num_rows;
          size_t m = host_mat_def_pos[i].num_cols;
          HostMatrix_array2d eigvects;
          HostVector_array1d eigvals;
          HostMatrix_array2d mat_OP1_copy;
          HostVector_array1d y1, eigvec(m);

          cusp::copy(host_mat_def_pos[i], mat_OP1_copy);
          cuspla::geev(mat_OP1_copy, eigvals, eigvects);

          for(size_t j=0; j<eigvals.size(); j++){
              thrust::copy(eigvects.values.begin()+ j*m, eigvects.values.begin()+ (j+1)*m,eigvec.begin());
              cuspla::gemv(host_mat_def_pos[i], eigvec, y1, false);
              cusp::blas::scal(eigvec, (ValueType)eigvals[j]);

              std::stringstream j_str, eigval_str;
              j_str << j;
              eigval_str << eigvals[j];

              ValueType errRel = nrmVector("host_GEEV eigval["+j_str.str()+"]:"+eigval_str.str()+" "+path_def_pos[i], y1, eigvec);
              CPPUNIT_ASSERT( errRel < 1.0e-2 );

          }
      }
  }

  void test_device_GEEV()
  {

      for(size_t i=0; i<path_def_pos.size(); i++){

          size_t n = host_mat_def_pos[i].num_rows;
          size_t m = host_mat_def_pos[i].num_cols;
          DeviceMatrix_array2d eigvects;
          DeviceVector_array1d eigvals;
          DeviceMatrix_array2d mat_OP1_copy;
          HostVector_array1d y1, eigvec(m);

          cusp::copy(dev_mat_def_pos[i], mat_OP1_copy);
          cuspla::geev(mat_OP1_copy, eigvals, eigvects);

          for(size_t j=0; j<eigvals.size(); j++){
              thrust::copy(eigvects.values.begin()+ j*m, eigvects.values.begin()+ (j+1)*m,eigvec.begin());
              cuspla::gemv(host_mat_def_pos[i], eigvec, y1, false);
              cusp::blas::scal(eigvec, (ValueType)eigvals[j]);

              std::stringstream j_str, eigval_str;
              j_str << j;
              eigval_str << eigvals[j];

              ValueType errRel = nrmVector("device_GEEV eigval["+j_str.str()+"]:"+eigval_str.str()+" "+path_def_pos[i], y1, eigvec);
              CPPUNIT_ASSERT( errRel < 1.0e-2 );

          }
      }
  }


  void test_host_GETRI()
  {

      for(size_t i=0; i<path_def_pos.size(); i++){
    	  HostMatrix_array2d A_inv;
    	  cusp::copy(host_mat_def_pos[i], A_inv);
    	  cuspla::getri(A_inv);
    	  cuspla::getri(A_inv);

          ValueType errRel = nrmVector("host_GETRIEV "+path_def_pos[i], A_inv.values, host_mat_def_pos[i].values);
          CPPUNIT_ASSERT( errRel < 1.0e-2 );

      }
  }


  void test_device_GETRI()
  {

      for(size_t i=0; i<path_def_pos.size(); i++){
    	  DeviceMatrix_array2d A_inv;
    	  cusp::copy(dev_mat_def_pos[i], A_inv);
    	  cuspla::getri(A_inv);
    	  cuspla::getri(A_inv);

    	  HostMatrix_array2d A_inv_host;
    	  cusp::copy(A_inv, A_inv_host);
          ValueType errRel = nrmVector("host_GETRIEV "+path_def_pos[i], A_inv_host.values, host_mat_def_pos[i].values);
          CPPUNIT_ASSERT( errRel < 1.0e-2 );

      }
  }




template <typename Array1d>
ValueType nrmVector(std::string title, Array1d& A, Array1d& A2){
      ValueType nrmA = cusp::blas::nrm2(A);
      ValueType nrmA2 = cusp::blas::nrm2(A2);
      // Calculates the difference and overwrite the matrix C
      cusp::blas::axpy(A, A2, ValueType(-1));
      ValueType nrmDiff = cusp::blas::nrm2(A2);



      ValueType errRel = ValueType(0);
      if(nrmA==ValueType(0))
          errRel = ValueType(1.0e-30);
      else
          errRel = nrmDiff/nrmA;

#ifdef VERBOSE
#ifndef VVERBOSE
      if(errRel != errRel || errRel >= 1.0e-2){ // Checks if error is nan
#endif VVERBOSE

        std::cout << title << ": AbsoluteErr=" << nrmDiff <<\
                " RelativeErr=" << errRel << "\n" << std::endl;
#ifndef VVERBOSE
      }
#endif VVERBOSE
#endif


      return errRel;
}





};





CPPUNIT_TEST_SUITE_REGISTRATION( CusplaTestCase );

int main(int argc, char** argv)
{

    CppUnit::TextUi::TestRunner runner;
    CppUnit::TestFactoryRegistry &registry = CppUnit::TestFactoryRegistry::getRegistry();
    runner.addTest( registry.makeTest() );
    runner.run();
    return 0;

}



